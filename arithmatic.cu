/* Author: Sidheswar Ghosh*/

#include "arithmatic.h"


int main(int argc, char** argv)
{   

    if( argc != 3)
    {
     printf("Usage: Please provide all below arguments...\n");
     printf("First Value\n");
     printf("Second Value\n");     
     return -1;
    } 

    int x_h = atoi(argv[1]);
    int y_h = atoi(argv[2]);   

    int *x_d, *y_d;
    float *res_h, *res_d;   

    //Allocate CUDA device memory
    hipMalloc(&x_d, sizeof(int));
    hipMalloc(&y_d, sizeof(int));
    hipMalloc(&res_d, sizeof(float));   

    //Copy value to CUDA device from host
    hipMemcpy(x_d, &x_h, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(y_d, &y_h, sizeof(int), hipMemcpyHostToDevice);    
    hipMemset(res_d, 0.0, sizeof(float));

    //Call CUDA function
    addition<<<1,1>>>(x_d, y_d, res_d);
    res_h = (float*)malloc(sizeof(float));
    hipMemcpy(res_h, res_d, sizeof(float), hipMemcpyDeviceToHost); //Copy result from CUDA device
    printf("Addition Result:%f\n", *res_h);

    //Call CUDA function
    substraction<<<1,1>>>(x_d, y_d, res_d);
    res_h = (float*)malloc(sizeof(float));
    hipMemcpy(res_h, res_d, sizeof(float), hipMemcpyDeviceToHost); //Copy result from CUDA device
    printf("Substraction Result:%f\n", *res_h);

    //Call CUDA function
    multiplication<<<1,1>>>(x_d, y_d, res_d);
    res_h = (float*)malloc(sizeof(float));
    hipMemcpy(res_h, res_d, sizeof(float), hipMemcpyDeviceToHost); //Copy result from CUDA device
    printf("Multiplication Result:%f\n", *res_h);

    //Call CUDA function
    division<<<1,1>>>(x_d, y_d, res_d);
    res_h = (float*)malloc(sizeof(float));
    hipMemcpy(res_h, res_d, sizeof(float), hipMemcpyDeviceToHost); //Copy result from CUDA device
    printf("Division Result:%f\n", *res_h);

    hipDeviceSynchronize();    
   
    return 0;
}